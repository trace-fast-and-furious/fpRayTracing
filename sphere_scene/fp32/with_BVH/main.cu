#include "hip/hip_runtime.h"
/*
 * ===================================================
 *
 *       Filename:  main.cu
 *    Description:  Ray Tracing In One Weekend (RTIOW): ~BVH 
 *        Created:  2022/07/13
 * 
 * ===================================================
 */


// Preprocessors

#include "moving_sphere.h"
#include "material.h"
#include "utility.h"
#include "color.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"
#include "bvh.h"

#include "mkPpm.h"
#include "mkCuda.h"
#include "mkClockMeasure.h"

#include <iostream>

#define MAX_SIZE 500

unsigned char *array;

// Functions


// 1. random_scene: Implements the 3D World.
hittable_list random_scene(int n) {
    	hittable_list world;
	int count = 0;
    
	auto ground_material = make_shared<lambertian>(color(0.5, 0.5, 0.5));
	//auto ground_material = make_shared<lambertian>(color(0.5, 0.5, 0.0));

    	world.add(make_shared<sphere>(++count, point3(0,-1000,0), 1000, ground_material));

    	for (int a = -n; a < n; a++) {
		for (int b = -n; b < n; b++) {

			// Generate constant scene primitives.
			float choose_mat = (a * 11 + b)/121;

			// Generate random scene primitives.
			//auto choose_mat = random_float();
	    		//point3 center(a + 0.9*random_float(), 0.2, b + 0.9*random_float());
	    		point3 center(a, 0.2, b);

	    		if ((center - point3(4, 0.2, 0)).length() > 0.9) {
			    	shared_ptr<material> sphere_material;

				if (choose_mat < 0.8) {
    		
					// diffuse
		    			auto albedo = color::random() * color::random();
		    			sphere_material = make_shared<lambertian>(albedo);
					auto center2 = center + vec3(0, random_float(0,.5), 0);
		    			world.add(make_shared<sphere>(
					   	++count, center, 0.2, sphere_material));
				} else if (choose_mat < 0.95) {
		    			// metal
		    			auto albedo = color::random(0.5, 1);
		    			auto fuzz = random_float(0, 0.5);
		    			sphere_material = make_shared<metal>(albedo, fuzz);
		    			world.add(make_shared<sphere>(++count, center, 0.2, sphere_material));
				} else {
		    			// glass
		    			sphere_material = make_shared<dielectric>(1.5);
		    			world.add(make_shared<sphere>(++count, center, 0.2, sphere_material));
				}
	    		}
		}
    	}

	auto material1 = make_shared<dielectric>(0.5);
	  world.add(make_shared<sphere>(++count, point3(0, 1, 0), 1.0, material1));

    	//world.add(make_shared<sphere>(++count, point3(3, 1, -0.5), 1.0, material1));

    	auto material2 = make_shared<lambertian>(color(0.4, 0.2, 0.1));
    	world.add(make_shared<sphere>(++count, point3(-4, 1, 0), 1.0, material2));

    	auto material3 = make_shared<metal>(color(0.7, 0.6, 0.5), 0.0);
    	world.add(make_shared<sphere>(++count, point3(4, 1, 0), 1.0, material3));
    //	world.add(make_shared<sphere>(++count, point3(3, 1, 2), 1.0, material3));

		auto material4 = make_shared<metal>(color(0.5, 0.7, 0.5), 0.1);
	//	world.add(make_shared<sphere>(++count, point3(3, 1, -0.5), 1.0, material4));


		auto material5 = make_shared<lambertian>(color(1.0, 0.0, 0.6));
	//	world.add(make_shared<sphere>(++count, point3(5, 0.5, 0), 0.5, material5));
		
		auto material6 = make_shared<dielectric>(0.5);
	//	world.add(make_shared<sphere>(++count, point3(5, 0.3, 1.3), 0.3, material6));

//	return world;
	
	// Constructing BVH
	hittable_list world_bvh;
	world_bvh.add(make_shared<bvh_node>(world, 0, 1));
	printf("\n\n================================== BVH CONSTURCTION COMPLETED ==================================\n\n\n");


	return world_bvh;
}



// 2. ray_color: calculates color of the current ray intersection point.
color ray_color(const ray& r, const hittable& world, int depth) {
    
	hit_record rec;



		// RT18: CHECK THE BACKGROUND COLOR
		//return vec3(0.5, 0.7, 1.0);


 
     	// Limit the number of child ray.
       	if (depth <= 0)
	       	return color(0, 0, 0);  // If the ray hits objects more than 'depth' times, consider that no light approaches the current point.

    	// If the ray hits an object: Hittable Object
    	if (world.hit(r, 0.001, infinity, rec)) {
		
		ray scattered;
		color attenuation;
		if (rec.mat_ptr->scatter(r, rec, attenuation, scattered))
	    		return attenuation * ray_color(scattered, world, depth-1);
		return color(0,0,0);
	}

    	// If the ray hits no object: Background
    	vec3 unit_direction = unit_vector(r.direction());
    	auto t = 0.5 * (unit_direction.y() + 1.0);
    	return (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
    	//return (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.7, 0.7, 1.0);

}



// 3. main
int main() {

	// Measure the execution time.
	mkClockMeasure *ckCpu = new mkClockMeasure("CPU CODE");
	ckCpu->clockReset();


    	// Image
	auto aspect_ratio = 16.0 / 9.0;
    	int image_width = 100;  //400
       	int samples_per_pixel = 5;    
	const int max_depth = 5;

	// Objects
	int n = 0;
	int object_num = (n+n)*(n+n)+4;

	
	ckCpu->clockResume();
    	// World
	hittable_list world = random_scene(n);

    	ckCpu->clockPause();
    	ckCpu->clockPrint("Creat World");


    	// Camera
	point3 lookfrom(13,2,3);
    	point3 lookat(0,0,0);
    	vec3 vup(0,1,0);
    	auto dist_to_focus = 20.0;
    	auto aperture = 0.1;
	int image_height = static_cast<int>(image_width / aspect_ratio);
    	camera cam(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus, 0.0, 1.0);


	// Rendered Image Array
	array = (unsigned char *)malloc(sizeof(unsigned char) * image_width * image_height * 3);


	ckCpu->clockReset();
	ckCpu->clockResume();


	// Render
	float r, g, b;

	// RT18
	//PRINT PIXEL VALUES OF THE OUTPUT IMAGE: printf("------------------- IMAGE -------------------\n");

	for (int j = 0; j < image_height; ++j) {
	   	for (int i = 0; i < image_width; ++i) {
	       		int idx = (j * image_width + i) * 3;
		  	color pixel_color(0, 0, 0);

				for (int s = 0; s < samples_per_pixel; ++s) {
			      	auto u = (i + random_float()) / (image_width - 1);
				auto v = ((image_height-j-1) + random_float()) / (image_height - 1);

				ray cur_ray = cam.get_ray(u, v);


				// RT17: FOR DEBUGGING
				/*
				printf("(RENDER) Pixel (%lf, %lf): Ray Direction = (%lf, %lf, %lf)\n\n", 
				u, v, 
				(cur_ray.direction()).e[0], (cur_ray.direction()).e[1], (cur_ray.direction()).e[2]);
				*/

				pixel_color += ray_color(cur_ray, world, max_depth);

				r = pixel_color.x();
				g = pixel_color.y();
				b = pixel_color.z();

				// Antialiasing
				float scale = 1.0 / samples_per_pixel;
				r = sqrt(scale * r);
				g = sqrt(scale * g);
				b = sqrt(scale * b);
				
				
//				printf("[%dx%d s:%d] %lf %lf %lf\n", j, i, s, pixel_color[0], pixel_color[1], pixel_color[2]);
				//printf("[%d] %f %f %f\n", s, pixel_color[0], pixel_color[1], pixel_color[2]);
			}
            
	    		array[idx] = (256 * clamp(r, 0.0, 0.999));
	    		array[idx+1] = (256 * clamp(g, 0.0, 0.999));
	    		array[idx+2] = (256 * clamp(b, 0.0, 0.999));

				// RT18 - PRINT PIXEL VALUES OF THE OUTPUT IMAGE:
				//printf("  R:%d, G:%d, B:%d\n", array[idx], array[idx+1], array[idx+2]);

    		}
    	}
		// RT18 - PRINT PIXEL VALUES OF THE OUTPUT IMAGE: 
		//printf("---------------------------------------------\n");


    	ckCpu->clockPause();
    	ckCpu->clockPrint("Rendering");

    	ppmSave("img.ppm", array, image_width, image_height, object_num, samples_per_pixel);

    	return 0;
}
